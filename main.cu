#include <string>
#include <chrono>

#define NDEBUG 1

#include <prover_reference_functions.hpp>

#include "multiexp/reduce.cu"

int main(int argc, char **argv) {
    mnt4753_libsnark::init_public_params();

    auto inputs = fopen(argv[2], "r");
    auto outputs = fopen(argv[3], "w");

    while (true) {
        size_t n;
        size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);

        if (elts_read == 0) { break; }

        size_t data_size = n * 2 * ELT_BYTES;

        auto x = allocate_memory(data_size);
        fread((void *)x.get(), data_size, 1, inputs);
        auto y = allocate_memory(data_size);
        fread((void *)y.get(), data_size, 1, inputs);
        auto out = allocate_memory(data_size);

        ec_multi<Fp2_MNT4>(x.get(), y.get(), out.get(), n);
        hipDeviceSynchronize();
        fwrite((void *)out.get(), data_size, 1, outputs);
    }
    return 0;
}
